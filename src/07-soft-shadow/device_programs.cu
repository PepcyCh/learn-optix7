#include "hip/hip_runtime.h"
#include "optix_device.h"

#include "shader_common.h"
#include "random.h"
#include "launch_params.h"

__constant__ LaunchParams optix_launch_params;

struct RayPayload {
    pcm::Vec3 color;
    RandomNumberGenerator rng;
};

struct VisibilityRayPayload {
    float visibility;
};

static __device__ void SampleLight(
    const pcm::Vec3 &pos,
    RandomNumberGenerator &rng,
    pcm::Vec3 &light_dir,
    pcm::Vec3 &light_strength,
    float &light_dist,
    float &sample_pdf
) {
    const auto &lights = optix_launch_params.light;

    float val = rng.NextFloat(0.0f, lights.light_count);
    uint32_t light_index = min(uint32_t(val), lights.light_count - 1);
    val -= light_index;
    if (val > lights.data[light_index].at_probability * lights.light_count) {
        light_index = lights.data[light_index].at_another_index;
    }

    const pcm::IVec3 ind = lights.data[light_index].index;
    const pcm::Vec3 p0 = lights.vertex[ind.X()];
    const pcm::Vec3 p1 = lights.vertex[ind.Y()];
    const pcm::Vec3 p2 = lights.vertex[ind.Z()];
    const pcm::Vec3 cross = (p1 - p0).Cross(p2 - p0);
    const pcm::Vec3 norm = cross.Normalize();

    const float r0 = rng.NextFloat(0.0f, 1.0f);
    const float r0_sqrt = sqrt(r0);
    const float r1 = rng.NextFloat(0.0f, 1.0f);

    const float u = 1.0f - r0_sqrt;
    const float v = r0_sqrt * (1.0f - r1);
    const float w = 1.0f - u - v;
    const pcm::Vec3 light_pos = u * p0 + v * p1 + w * p2;

    const pcm::Vec3 light_vec = light_pos - pos;
    const float light_dist_sqr = light_vec.MagnitudeSqr();
    light_dist = sqrt(light_dist_sqr);
    light_dir = light_vec / light_dist;
    light_strength = lights.data[light_index].strength * max(norm.Dot(-light_dir), 0.0) / light_dist_sqr;

    sample_pdf = lights.data[light_index].at_probability / (cross.Length() * 0.5f);
}

OPTIX_CLOSESTHIT(Radiance)() {
    RayPayload *payload = GetRayPayload<RayPayload>();
    const TriMeshData *data = (const TriMeshData *)optixGetSbtDataPointer();
    const uint32_t prim_id = optixGetPrimitiveIndex();

    const uint32_t i0 = data->index[prim_id * 3];
    const uint32_t i1 = data->index[prim_id * 3 + 1];
    const uint32_t i2 = data->index[prim_id * 3 + 2];

    const float bc_u = optixGetTriangleBarycentrics().x;
    const float bc_v = optixGetTriangleBarycentrics().y;
    const float bc_w = 1.0f - bc_u - bc_v;

    const pcm::Vec3 pos = bc_w * data->vertex[i0] + bc_u * data->vertex[i1] + bc_v * data->vertex[i2];
    const pcm::Vec3 norm = (bc_w * data->normal[i0] + bc_u * data->normal[i1] + bc_v * data->normal[i2]).Normalize();

    pcm::Vec3 base_color = data->base_color;
    if (data->base_color_mapped) {
        const pcm::Vec2 uv = bc_w * data->uv[i0] + bc_u * data->uv[i1] + bc_v * data->uv[i2];
        float4 tex_val = tex2D<float4>(data->base_color_map, uv.X(), uv.Y());
        base_color *= pcm::Vec3(tex_val.x, tex_val.y, tex_val.z);
    }

    pcm::Vec3 light_dir;
    pcm::Vec3 light_strength;
    float light_dist;
    float light_pdf;
    SampleLight(pos, payload->rng, light_dir, light_strength, light_dist, light_pdf);

    const float dot = max(norm.Dot(light_dir), 0.0f);

    VisibilityRayPayload shadow_payload;
    shadow_payload.visibility = 0.0f;

    RayDesc ray;
    ray.origin = pos;
    ray.direction = light_dir;
    ray.t_min = 0.001f;
    ray.t_max = light_dist - 0.001f;

    TraceRay(
        optix_launch_params.traversable,
        OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
        0xff,
        static_cast<uint32_t>(RayType::eShadow),
        static_cast<uint32_t>(RayType::eCount),
        static_cast<uint32_t>(RayType::eShadow),
        ray,
        &shadow_payload
    );

    payload->color = dot * shadow_payload.visibility * base_color * light_strength / light_pdf + 0.1f * base_color;
}

OPTIX_CLOSESTHIT(Empty)() {}

OPTIX_ANYHIT(Empty)() {}

OPTIX_MISS(Radiance)() {
    RayPayload *payload = GetRayPayload<RayPayload>();
    payload->color = pcm::Vec3(1.0f, 1.0f, 1.0f);
}

OPTIX_MISS(Shadow)() {
    VisibilityRayPayload *payload = GetRayPayload<VisibilityRayPayload>();
    payload->visibility = 1.0f;
}

OPTIX_RAYGEN(RenderFrame)() {
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    const auto &fr = optix_launch_params.frame;
    const auto &cam = optix_launch_params.camera;

    const uint32_t buffer_index = ix + iy * fr.width;
    RayPayload payload;
    payload.color = pcm::Vec3::Zero();
    payload.rng.Seed(buffer_index + fr.curr_time);

    const float uu = payload.rng.NextFloat(0.0f, 1.0f);
    const float vv = payload.rng.NextFloat(0.0f, 1.0f);
    const float u = (ix + uu) / fr.width - 0.5f;
    const float v = 0.5f - (iy + vv) / fr.height;

    RayDesc ray;
    ray.origin = cam.position;
    ray.direction = (cam.direction + cam.right * u + cam.up * v).Normalize();
    ray.t_min = 0.0f;
    ray.t_max = 1e20f;

    TraceRay(
        optix_launch_params.traversable,
        OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        0xff,
        static_cast<uint32_t>(RayType::eRadiance),
        static_cast<uint32_t>(RayType::eCount),
        static_cast<uint32_t>(RayType::eRadiance),
        ray,
        &payload
    );

    const pcm::Vec4 accum_result = fr.accum_weight * pcm::Vec4(payload.color, 1.0f)
        + (1.0f - fr.accum_weight) * fr.prev_color_buffer[buffer_index];
    fr.color_buffer[buffer_index] = accum_result;
}